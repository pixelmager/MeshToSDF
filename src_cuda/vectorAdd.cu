#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <assert.h>

#define WIN32_LEAN_AND_MEAN
#define NOMINMAX
#include <Windows.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
///__global__ void
///vectorAdd(const float *A, const float *B, float *C, int numElements)
///{
///    int i = blockDim.x * blockIdx.x + threadIdx.x;
///
///    if (i < numElements)
///    {
///        C[i] = A[i] + B[i];
///    }
///}

// ==================================================================
//note: reference https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
//                https://docs.nvidia.com/nsight-visual-studio-edition/cuda-debugger/
//                https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/

typedef float float32_t;

//note: from https://randomascii.wordpress.com/2012/02/25/comparing-floating-point-numbers-2012-edition/
bool AlmostEqualRelative(float A, float B, float maxRelDiff = FLT_EPSILON)
{
	// Calculate the difference.
	float diff = fabs(A - B);
	A = fabs(A);
	B = fabs(B);
	// Find the largest
	float largest = (B > A) ? B : A;

	if (diff <= largest * maxRelDiff)
		return true;
	return false;
}


LARGE_INTEGER currentTime;
LARGE_INTEGER m_timer_frequency;
void init_timers()
{
	QueryPerformanceFrequency( &m_timer_frequency );
}
uint64_t gettime_ms()
{
	QueryPerformanceCounter(&currentTime);
	return static_cast<uint64_t>( static_cast<double>(currentTime.QuadPart) / static_cast<double>(m_timer_frequency.QuadPart) * 1000.0 );
}

// =================================================

struct vec3_t
{
	float32_t x, y, z;
	//vec3_t(){}
	__host__ __device__ vec3_t( float32_t in_x, float32_t in_y, float32_t in_z ) : x(in_x), y(in_y), z(in_z) {}
	__host__ __device__ vec3_t( const vec3_t &v ) : x(v.x), y(v.y), z(v.z) {}
	//explicit vec3_t( __m128 v ) : x(v.m128_f32[0]), y(v.m128_f32[1]), z(v.m128_f32[2] ) {}
};

__host__ __device__
float32_t dot( const vec3_t &a, const vec3_t &b )
{
	return a.x*b.x + a.y*b.y + a.z*b.z;
}
__host__ __device__
vec3_t operator+( const vec3_t &a, const vec3_t &b )
{
	return vec3_t( a.x + b.x,
				   a.y + b.y,
				   a.z + b.z );
}
__host__ __device__
vec3_t sub( const vec3_t &a, const vec3_t &b )
{
	return vec3_t( a.x - b.x,
				a.y - b.y,
				a.z - b.z );
}
__host__ __device__
vec3_t operator-( const vec3_t &a, const vec3_t &b )
{
	return sub(a, b);
}
__host__ __device__
vec3_t mul( const vec3_t &a, const vec3_t &b )
{
	return vec3_t(  a.x * b.x,
					a.y * b.y,
					a.z * b.z );
}
__host__ __device__
vec3_t operator*( const vec3_t &a, const vec3_t &b )
{
	return mul(a, b);
}
__host__ __device__
vec3_t operator*( const vec3_t &v, const float32_t c )
{
	return vec3_t( v.x * c,
				   v.y * c,
				   v.z * c );
}
__host__ __device__
vec3_t operator*( const float32_t c, const vec3_t &v )
{
	return vec3_t( v.x * c,
				   v.y * c,
				   v.z * c );
}
__host__ __device__
vec3_t cross( const vec3_t &a, const vec3_t &b )
{
	return vec3_t( a.y*b.z - a.z*b.y,
				   a.z*b.x - a.x*b.z,
				   a.x*b.y - a.y*b.x );
}
__host__ __device__
vec3_t min(const vec3_t& v0, const vec3_t& v1)
{
	return vec3_t(min(v0.x, v1.x),
		min(v0.y, v1.y),
		min(v0.z, v1.z));
}
__host__ __device__
vec3_t max(const vec3_t& v0, const vec3_t& v1)
{
	return vec3_t(max(v0.x, v1.x),
		max(v0.y, v1.y),
		max(v0.z, v1.z));
}


// =================================================

__host__ __device__
float32_t sign( const float32_t v )
{
	//TODO: return sign bit?
	//return (v>=0.0f) ? 1.0f : -1.0f;

	// https://www.khronos.org/registry/OpenGL-Refpages/gl4/html/sign.xhtml
	if ( v == 0.0f )
		return 0.0f;
	return (v>0.0f) ? 1.0f : -1.0f;
}

__host__ __device__
float32_t clamp( float32_t v, float32_t mn, float32_t mx )
{
	return max(mn, min(mx, v));
}

// =================================================

struct sdf_t
{
	struct sdfheader_t
	{
			int32_t dim_x, dim_y, dim_z;
			float32_t bb_mn_x, bb_mn_y, bb_mn_z;
			float32_t bb_mx_x, bb_mx_y, bb_mx_z;
	} header;
	
    float32_t *d_data;
	float32_t *h_data;
};
struct aabb_t
{
	vec3_t mn;
	vec3_t mx;
	aabb_t() : mn(FLT_MAX,FLT_MAX,FLT_MAX), mx(-FLT_MAX,-FLT_MAX,-FLT_MAX) {}
};

void init_sdf( sdf_t *sdf, aabb_t bb, int32_t siz_x, int32_t siz_y, int32_t siz_z )
{
	//enum { SIMD_SIZ = 16, SIMD_ALIGN=4*SIMD_SIZ };
	//const int32_t simd_align = sizeof(float32_t) * simd_siz;

	sdf->header.dim_x = siz_x;
	sdf->header.dim_y = siz_y;
	sdf->header.dim_z = siz_z;

	//note: extend bb by a border
	const int32_t border_siz = sdf->header.dim_x/4;
	const vec3_t bb_gridcell_siz = (bb.mx - bb.mn) * vec3_t( 1.0f/sdf->header.dim_x, 1.0f/sdf->header.dim_y, 1.0f/sdf->header.dim_z );
	bb.mn = bb.mn - (float32_t)border_siz * bb_gridcell_siz;
	bb.mx = bb.mx + (float32_t)border_siz * bb_gridcell_siz;

	sdf->header.bb_mn_x = bb.mn.x;
	sdf->header.bb_mn_y = bb.mn.y;
	sdf->header.bb_mn_z = bb.mn.z;

	sdf->header.bb_mx_x = bb.mx.x;
	sdf->header.bb_mx_y = bb.mx.y;
	sdf->header.bb_mx_z = bb.mx.z;

	sdf->h_data = (float32_t*)_aligned_malloc( sizeof(float32_t) * sdf->header.dim_x * sdf->header.dim_y * sdf->header.dim_z, 16 );

    sdf->d_data = NULL;
    hipError_t err = hipMalloc((void **)&sdf->d_data, sizeof(float32_t) * sdf->header.dim_x * sdf->header.dim_y * sdf->header.dim_z );
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector sdf-data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

void deinit_sdf( sdf_t *sdf )
{
	_aligned_free( sdf->h_data );
    hipFree(sdf->d_data );
}


// =================================================


//note: host, fileread
struct itm_header_t
{
	uint64_t num_indices;
	uint64_t num_positions;
};
//uint32_t indices[num_indices];
//float32_t positions[num_positions];

struct itm_t
{
	itm_header_t header;
	uint32_t *indices;
	float32_t *positions;
};

itm_t* itm_readgeom( char const * const fn )
{
	FILE *infile;
	errno_t err = fopen_s( &infile, fn, "rb" );
	if ( infile == nullptr ) { printf("wtf2\n%i\n", err); return nullptr; }

	itm_t * const ret = (itm_t*)malloc( sizeof(itm_t) );

	fread( &ret->header, sizeof(itm_header_t), 1, infile );

	ret->indices = (uint32_t*)malloc( sizeof(uint32_t) * ret->header.num_indices );
	ret->positions = (float32_t*)malloc( sizeof(float32_t) * ret->header.num_positions );

	fread( ret->indices, sizeof(uint32_t), ret->header.num_indices, infile );
	fread( ret->positions, sizeof(float32_t), ret->header.num_positions, infile );
	fclose( infile );

	return ret;
}

//note: from http://iquilezles.org/www/articles/triangledistance/triangledistance.htm
__host__ __device__
float32_t dot2( const vec3_t &v ) { return dot(v,v); }

__host__ __device__
float32_t udTriangle_sq( const vec3_t &v1, const vec3_t &v2, const vec3_t &v3, const vec3_t &p )
{
    vec3_t v21 = v2 - v1; vec3_t p1 = p - v1;
    vec3_t v32 = v3 - v2; vec3_t p2 = p - v2;
    vec3_t v13 = v1 - v3; vec3_t p3 = p - v3;
    vec3_t nor = cross( v21, v13 );

    //return sqrtf((sign(dot(cross(v21,nor),p1)) + 
	return ((sign(dot(cross(v21,nor),p1)) + 
             sign(dot(cross(v32,nor),p2)) + 
             sign(dot(cross(v13,nor),p3))<2.0f) 
             ?
             min( min( 
             dot2(v21*clamp(dot(v21,p1)/dot2(v21),0.0f,1.0f)-p1),
             dot2(v32*clamp(dot(v32,p2)/dot2(v32),0.0f,1.0f)-p2) ),
             dot2(v13*clamp(dot(v13,p3)/dot2(v13),0.0f,1.0f)-p3) )
             :
             dot(nor,p1)*dot(nor,p1)/dot2(nor) );
}

// =================================================

//note: naive, map every thread to a gridcell, run through all triangles per cell

__host__ __device__
float32_t calc_gridcell( const int32_t num_indices, uint32_t const * const tri_indices, const vec3_t *positions,
						 int32_t x, int32_t y, int32_t z, int32_t xn, int32_t yn, int32_t zn,
						 const vec3_t &bb_mn, const vec3_t &bb_range )
{
	vec3_t p_nm = vec3_t( (static_cast<float32_t>(x)+0.5f) / static_cast<float32_t>(xn),
						  (static_cast<float32_t>(y)+0.5f) / static_cast<float32_t>(yn),
						  (static_cast<float32_t>(z)+0.5f) / static_cast<float32_t>(zn) );
	vec3_t p = bb_mn + bb_range * p_nm;

	float d_min = FLT_MAX;
	for ( size_t idx_tri=0,num_tris=num_indices/3; idx_tri<num_tris; ++idx_tri )
	{
		const uint32_t idx0 = tri_indices[ 3*idx_tri+0 ];
		const uint32_t idx1 = tri_indices[ 3*idx_tri+1 ];
		const uint32_t idx2 = tri_indices[ 3*idx_tri+2 ];
		const vec3_t &p0 = positions[ idx0 ];
		const vec3_t &p1 = positions[ idx1 ];
		const vec3_t &p2 = positions[ idx2 ];

		float32_t ud = udTriangle_sq( p0, p1, p2, p );
		if ( ud < d_min )
			d_min = ud;
	}

	return d_min;
}

__global__
void sdf_naive( sdf_t::sdfheader_t sdfheader, const itm_header_t itmheader, const uint32_t *tri_indices, const float32_t *mesh_positions, float32_t *out_sdf_data)
{
	vec3_t const * const positions = reinterpret_cast<vec3_t const * const>( &mesh_positions[0] );

	const int32_t x = threadIdx.x;
	const int32_t y = threadIdx.y;
	const int32_t z = threadIdx.z;
	
	const int32_t xn = sdfheader.dim_x;
	const int32_t yn = sdfheader.dim_y;
	const int32_t zn = sdfheader.dim_z;

	const vec3_t bb_min = vec3_t( sdfheader.bb_mn_x, sdfheader.bb_mn_y, sdfheader.bb_mn_z );
	const vec3_t bb_max = vec3_t( sdfheader.bb_mx_x, sdfheader.bb_mx_y, sdfheader.bb_mx_z );
	const vec3_t bb_range = bb_max-bb_min;

	float32_t d_min = calc_gridcell( itmheader.num_indices, tri_indices, positions,
									 x, y, z, xn, yn, zn,
		                             bb_min, bb_range);
		
    int idx = x + y*xn + z*xn*yn;
	out_sdf_data[idx] = sqrtf( d_min );
}


void invoke_naive( const itm_t &mesh, sdf_t &sdf )
{
    const int numBlocks = 1;
    dim3 threadsPerBlock( sdf.header.dim_x, sdf.header.dim_y, sdf.header.dim_z );

    uint32_t *d_indices = NULL;
    checkCudaErrors( hipMalloc( &d_indices, sizeof(float32_t) * mesh.header.num_indices) );
    checkCudaErrors( hipMemcpy(d_indices, mesh.indices, sizeof(float32_t)*mesh.header.num_indices, hipMemcpyKind::hipMemcpyHostToDevice ) );

    float32_t *d_positions = NULL;
    checkCudaErrors( hipMalloc( &d_positions, sizeof(float32_t) * mesh.header.num_positions) );
    checkCudaErrors( hipMemcpy(d_positions, mesh.positions, sizeof(float32_t)*mesh.header.num_positions, hipMemcpyKind::hipMemcpyHostToDevice ) );

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
    sdf_naive<<<numBlocks, threadsPerBlock>>>(sdf.header, mesh.header, d_indices, d_positions, sdf.d_data);
	hipEventRecord(stop);


    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	checkCudaErrors( hipDeviceSynchronize() );
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf( "kernel-time %fms\n", milliseconds );
}

// =================================================


// ====

void eval_sdf__bruteforce( sdf_t &sdf, itm_t const * const mesh, float32_t *out_sdf_data )
{
	//printf("%s\n", __FUNCTION__);
	assert( mesh->header.num_positions % 3 == 0 );
	uint32_t const * const indices = reinterpret_cast<uint32_t const * const>( &mesh->indices[0] );
	vec3_t const * const positions = reinterpret_cast<vec3_t const * const>( &mesh->positions[0] );

	const vec3_t bb_min = vec3_t( sdf.header.bb_mn_x, sdf.header.bb_mn_y, sdf.header.bb_mn_z );
	const vec3_t bb_max = vec3_t( sdf.header.bb_mx_x, sdf.header.bb_mx_y, sdf.header.bb_mx_z );
	const vec3_t bb_range = bb_max-bb_min;

	for ( int z=0,zn=sdf.header.dim_z; z<zn; ++z ) {
	printf( "z=%d\n", z );
	for ( int y=0,yn=sdf.header.dim_y; y<yn; ++y ) {
	for ( int x=0,xn=sdf.header.dim_x; x<xn; ++x )
	{
		float32_t d_min = calc_gridcell( mesh->header.num_indices, indices, positions,
										 x, y, z, xn, yn, zn,
										 bb_min, bb_range );

		int idx = x + y*xn + z*xn*yn;
		out_sdf_data[idx] = sqrtf( d_min );
	}}}
}


// ==================================================================

/**
 * Host main routine
 */
int main(void)
{

    // =====================================================
    itm_t * mesh = itm_readgeom( "data/bunny.itm" );

	aabb_t bb;
	for ( size_t i=0, in=mesh->header.num_positions/3; i<in; ++i )
	{
		vec3_t *p = reinterpret_cast<vec3_t*>( &mesh->positions[ i ] );
		bb.mn = min( bb.mn, *p );
		bb.mx = max( bb.mx, *p );
	}

	//TODO: max 1024 all together...
    enum { GRID_SIZ_X=10,
           GRID_SIZ_Y=10,
           GRID_SIZ_Z=10
    };
    sdf_t sdf;    
    init_sdf(&sdf, bb, GRID_SIZ_X, GRID_SIZ_Y, GRID_SIZ_Z );


    // =====================================================

    invoke_naive( *mesh, sdf );


	float32_t *h_sdf = (float32_t*)malloc( sizeof(float32_t) * sdf.header.dim_x * sdf.header.dim_y * sdf.header.dim_z );
    checkCudaErrors( hipMemcpy(h_sdf, sdf.d_data, sizeof(float32_t) * sdf.header.dim_x * sdf.header.dim_y * sdf.header.dim_z, hipMemcpyDeviceToHost) );

	//for ( int i=0, n=sdf.header.dim_x * sdf.header.dim_y * sdf.header.dim_z; i<n; ++i )
	//{
	//	printf( "%f\n", h_sdf[i] );
	//}

	init_timers();
	uint64_t t0_ms = gettime_ms();
	eval_sdf__bruteforce( sdf, mesh, sdf.h_data );
	uint64_t t1_ms = gettime_ms();
	printf( "%dms\n", (int)(t1_ms-t0_ms) );

	bool sane = true;
	float32_t mindiff =  FLT_MAX;
	float32_t maxdiff = -FLT_MAX;
	for (int i = 0, n = sdf.header.dim_x * sdf.header.dim_y * sdf.header.dim_z; i < n; ++i)
	{
		const float32_t d0 = sdf.h_data[i];
		const float32_t d1 = h_sdf[i];
		if (!AlmostEqualRelative(d0, d1))
		{
			float32_t diff = abs(d1 - d0);
			if (diff < mindiff) mindiff = diff;
			if (diff > maxdiff) maxdiff = diff;
			sane = false;
		}
	}

	if( mindiff < maxdiff )
		printf("delta: [min;max]=[%f;%f]", mindiff, maxdiff);

	printf( "\nsane: %s\n", sane?"true":"false" );

    {
        free( mesh->indices );
        free( mesh->positions );
        free( mesh );

		//TODO: free device memory...
    }

    printf("Done\n");
    return 0;
}

